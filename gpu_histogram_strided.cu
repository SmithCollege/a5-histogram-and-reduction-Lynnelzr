#include <iostream>
#include <hip/hip_runtime.h>

__global__ void gpuHistogramStrided(int* input, int* histogram, int n, int numBins) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        int bin = input[i] % numBins;
        atomicAdd(&histogram[bin], 1);
    }
}

int main() {
    int numBins = 10;
    int *d_data, *d_histogram;

    // Loop over different array sizes (powers of 2)
    for (int arraySize = 1024; arraySize <= 1048576; arraySize *= 2) {
        // Allocate host memory
        int* h_data = new int[arraySize];
        int* h_histogram = new int[numBins]();

        // Initialize data on the host
        for (int i = 0; i < arraySize; ++i) {
            h_data[i] = i % numBins;
        }

        // Allocate device memory
        hipMalloc(&d_data, arraySize * sizeof(int));
        hipMalloc(&d_histogram, numBins * sizeof(int));

        // Copy data to device and initialize histogram on device
        hipMemcpy(d_data, h_data, arraySize * sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_histogram, 0, numBins * sizeof(int));

        // Create CUDA events for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Start timing
        hipEventRecord(start, 0);

        // Launch kernel
        gpuHistogramStrided<<<(arraySize + 255) / 256, 256>>>(d_data, d_histogram, arraySize, numBins);

        // Stop timing
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // Calculate elapsed time in milliseconds
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Copy result back to host
        hipMemcpy(h_histogram, d_histogram, numBins * sizeof(int), hipMemcpyDeviceToHost);

        // Print results for this array size
        std::cout << "Array Size: " << arraySize << "\nHistogram:\n";
        for (int i = 0; i < numBins; ++i) {
            std::cout << "Bin " << i << ": " << h_histogram[i] << std::endl;
        }
        std::cout << "GPU Runtime: " << milliseconds << " ms\n";
        std::cout << "----------------------------------------\n";

        // Free allocated memory
        delete[] h_data;
        delete[] h_histogram;
        hipFree(d_data);
        hipFree(d_histogram);

        // Destroy CUDA events
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    return 0;
}

